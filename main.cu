#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <chrono>
#include <getopt.h>
#include <sstream>
#include <cstdlib>

#include "kmeansgpu.cuh"

#define DIM 3
#define COMPARISON_ERROR 0.1f

template<unsigned int n>
float* readObjectsFromFile(std::string filepath, int* N)
{
    std::ifstream fileStream(filepath);
    if(!fileStream.good())
    {
        return NULL;
    }
    std::string input, number;
    getline(fileStream, input);
    *N = stoi(input);
    auto objects = new float[n * (*N)];
    int index = 0;
    while (getline(fileStream, input))
    {
        std::istringstream stream(input);
        while(getline(stream, number, ' ')) {
            objects[index++] = stof(number);
        }
    }
    return objects;
}

template<unsigned int n>
void writeResultsToFile(const char* membershipFilePath, const char* centersFilePath, int N, int k, int* membership, float* centers)
{
    std::ofstream membershipFileStream(membershipFilePath, std::ofstream::out | std::ofstream::trunc);
    if(!membershipFileStream.good())
    {
        std::cout << "kekium";
        return;
    }
    for(int i = 0; i < N; ++i)
    {
        membershipFileStream << membership[i] << std::endl;
    }
    membershipFileStream.close();
    std::ofstream centersFileStream(centersFilePath, std::ofstream::out | std::ofstream::trunc);
    if(!centersFileStream.good())
    {
        return;
    }
    for(int i = 0; i < k; ++i)
    {
        for(int j = 0; j < n; ++j)
        {
            centersFileStream << centers[i * n + j] << " ";
        }
        centersFileStream << std::endl;
    }
    centersFileStream.close();
}

template<unsigned int n>
float* generateRandomData(int N, int seed = 1234)
{
    srand(seed);

    float* data = new float[N * n];
    for(int i = 0; i < N; ++i)
    {
        for(int j = 0; j < n; ++j)
        {
            data[i * n + j] = -1000 + (rand() % 2000);
        }
    }

    return data;
}

template<unsigned int n>
void calculateAverageDistance(const float* objects, const float* centers, const int* membership, int N)
{
    float sum = 0;
    for(int i = 0; i < N; ++i)
    {
        sum += sqrt(distanceSquared<n>(objects + i * n, centers + membership[i] * n));
    }
    std::cout << "Average object to center distance is " << sum / N << std::endl;
}

void usage()
{
    std::cout << "Usage:" << std::endl;
    std::cout << "  kmeans.out [-f filepath | -n N] [options] k " << std::endl;
    std::cout << std::endl;
    std::cout << "Options:" << std::endl;
    std::cout << "  -c, --cpu-only      only run cpu algorithm" << std::endl;
    std::cout << "  -f, --file          specify a path to a file with data" << std::endl;
    std::cout << "  -g, --gpu-only      only run gpu algorithm" << std::endl;
    std::cout << "  -n, --generate      generate a random set of N objects" << std::endl;
    std::cout << "  -1                  use the first gpu algorithm" << std::endl;
    std::cout << "  -2                  use the second gpu algorithm" << std::endl;
    exit(EXIT_FAILURE);
}

int main(int argc, char** argv)
{
    int c;
    bool isCpuOnly = false;
    bool isGpuOnly = false;
    bool isFile = false;
    bool isGenerate = false;
    bool isDebug = false;
    bool isFirstGpu = false;
    static struct option long_options[] = {
        {"cpu-only", no_argument, NULL, 'c'},
        {"file", required_argument, NULL, 'f'},
        {"gpu-only", no_argument, NULL, 'g'},
        {"generate", required_argument, NULL, 'n'},
        {"debug", no_argument, NULL, 'd'},
        { NULL, 0, NULL, 0 }
    };
    std::string filepath;
    int N = 0;

    while (1)
    {
        c = getopt_long(argc, argv, "cdf:gn:12", long_options, NULL);
        if(c == -1)
            break;

        switch(c)
        {
            case '1':
                isFirstGpu = true;
                break;
            case '2':
                isFirstGpu = false;
                break;
            case 'c':
                isCpuOnly = true;
                break;
            case 'd':
                isDebug = true;
                break;
            case 'f':
                isFile = true;
                filepath = optarg;
                break;
            case 'g':
                isGpuOnly = true;
                break;
            case 'n':
                isGenerate = true;
                N = atoi(optarg);
                if(N < 1)
                {
                    usage();
                }
                break;
            default:
                usage();
                break;
        }
    }

    if (optind != argc - 1 || (isFile && isGenerate) || !(isFile || isGenerate)) {
        usage();
    }
    int k = atoi(argv[optind++]);

    if (isCpuOnly && isGpuOnly)
    {
        std::cout << "The -c and -g flags are mutually exclusive" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    // initialize data
    float* objects = 0;
    if (isFile)
    {
        objects = readObjectsFromFile<DIM>(filepath, &N);
    }
    else
    {
        objects = generateRandomData<DIM>(N);
    }

    float* cpuCenters, *gpuCenters;
    int* cpuMembership, *gpuMembership;
    if (!isGpuOnly)
    {
        std::cout << std::endl;
        std::cout << "Solving kmeans cpu..." << std::endl;
        auto start = std::chrono::high_resolution_clock::now();
        cpuMembership = kmeansCpu<DIM>(objects, N, k, &cpuCenters, isDebug);
        auto stop = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
        std::cout << "Total time for cpu: " << duration.count() << " microseconds" << std::endl;

        calculateAverageDistance<DIM>(objects, cpuCenters, cpuMembership, N);

        std::cout << "Writing cpu results to files results/cpu.membership and results/cpu.centers" << std::endl;
        writeResultsToFile<DIM>("results/cpu.membership", "results/cpu.centers", N, k, cpuMembership, cpuCenters);
    }

    if (!isCpuOnly)
    {
        std::cout << std::endl;
        std::cout << "Solving kmeans gpu..." << std::endl;
        auto start = std::chrono::high_resolution_clock::now();
        if (isFirstGpu)
        {
            gpuMembership = kmeansGpu<DIM>(objects, N, k, &gpuCenters, isDebug);
        }
        else
        {
            gpuMembership = kmeansGpu2<DIM>(objects, N, k, &gpuCenters, isDebug);
        }
        auto stop = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
        std::cout << "Total time for gpu: " << duration.count() << " microseconds" << std::endl;

        calculateAverageDistance<DIM>(objects, gpuCenters, gpuMembership, N);

        std::cout << "Writing gpu results to files results/gpu.membership and results/gpu.centers" << std::endl;
        writeResultsToFile<DIM>("results/gpu.membership", "results/gpu.centers", N, k, gpuMembership, gpuCenters);
    }

    std::cout << std::endl;
    std::cout << "Deleting objects" << std::endl;
    delete[] objects;
    if(!isGpuOnly)
    {
        delete[] cpuMembership;
        delete[] cpuCenters;
    }
    if(!isCpuOnly)
    {
        delete[] gpuMembership;
        delete[] gpuCenters;
    }
    return 0;
}
